
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hola_cuda(){
      printf("Hello World from GPU!\n");
}

int main() {
      //no hace absolutamente nada, pero no deberia dar error.
      hola_cuda<<<1,1>>>();
 
      return 0;
}
// Compilación:
// $ nvcc hola_cuda.cu
// Ejecución:
// $ ./a.out
